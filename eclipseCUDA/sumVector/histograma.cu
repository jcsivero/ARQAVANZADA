#include <stdio.h>
#include <time.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>


/**
 * CUDA Kernel Device code
 * Calcula el histograma de un vector pasado. 
 * Versión directa con operaciones atómicas a vector H en memoria de video
 */
__global__ void
histogram(int *V, int * H, int numElementsV, int numElementsH)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
	
if (i < numElementsV)
  {
	int index = V[i] % numElementsH;
	atomicAdd((H +index), 1); 
  }
	
}


/**
 * CUDA Kernel Device code
 * Calcula el histograma de un vector pasado.
 * Versión directa con operaciones atómicas a memoria compartida por bloque y un solo acceso atómico final a la memoria global de la CPU, tras sincronización de los hilos
 * con el resultado a memorida de video.
 */
 __global__ void
 histogramShared(int *V, int * H, int numElementsV, int numElementsH)
 {
 __shared__ int acc[8];  ///tamaño máximo del vector H.Ver si se puede optimizar para  crear en tiempo de ejecución el vector dentro del kernel.
 ///además, no se como inicializar la variable a cero inicialmente.

  int i = blockDim.x * blockIdx.x + threadIdx.x;
   
  if (i % blockDim.x == 0)  ///si es el hilo del principio de un bloque, me encargaré de inicializar a cero la variable contador y después de ser el hilo que escriba a memoria global de la GPU.
    for (int j=0; j < 8; j++)
      acc[j] = 0;

     __syncthreads();    

 if (i < numElementsV)
   {
     int index = V[i] % numElementsH;
     atomicAdd((&acc[0] + index), 1); 
     
   }
   __syncthreads();    

   if (i % blockDim.x == 0)    
     for (int j = 0; j < numElementsH; j++)     
        atomicAdd((H + j), acc[j]); 
 
   
 }

/**
 * CUDA Kernel Device code
 * Suma por reducción de los elementos de un vector
 */
 __global__ void
 sumHistogram(int * h, int blocksPerGrid) ///blocksPerGrid dice el número de bloques  que realmente habría, o sea, los reales multiplicados por 2.
 {
     ///blockDim equivale al tamño del histograma, ya que se creó el bloque con tantos hilos como tamaño del histograma.

     int i = blockDim.x * blockIdx.x + threadIdx.x; ///posicion dentro del vector del histograma.
     int j = (blocksPerGrid -1 - blockIdx.x) * blockDim.x + threadIdx.x; ///valor  que voy a sumar dentro del histograma  que le corresponde. 
     ///O sea, el primer bloque(equivalente al tamaño  de elementos del histograma) se sumará con el último, el segundo, con el penúltimo.....

    h[i] = h[i] + h[j];
    
 }

/**
 * CUDA Kernel Device code
 * Calcula el histograma de un vector pasado.
 * Versión directa con operaciones atómicas a memoria de video, pero con un histograma por bloque.
 * Se sigue utilizando un vector H, pero su tamaño es numElementsH * número de bloques.
 * O sea, hay un histograma de tamaño numElementsH por cada bloque y su ubicación en memoria es como vector, un histograma seguido del otro
 */
__global__ void
histogramByBlock(int *V, int * H, int numElementsV, int numElementsH)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
	
if (i < numElementsV)
  {	
	int index = (V[i] % numElementsH) + (blockIdx.x * numElementsH); ///me posiciono en histograma asociado a este bloque y en la posición correspondiente
	atomicAdd((H +index), 1); 
  }
	
}

/**
 * CUDA Kernel Device code
 * Calcula el histograma de un vector pasado.
 * Versión directa con operaciones atómicas a memoria de video, pero con un histograma por bloque.
 * Se sigue utilizando un vector H, pero su tamaño es numElementsH * número de bloques.
 * O sea, hay un histograma de tamaño numElementsH por cada bloque y su ubicación en memoria es como vector, un histograma seguido del otro
 */
 __global__ void
 histogramByBlockShared(int *V, int * H, int numElementsV, int numElementsH)
 {
     int i = blockDim.x * blockIdx.x + threadIdx.x;
     
 if (i < numElementsV)
   {	
     int index = (V[i] % numElementsH) + (blockIdx.x * numElementsH); ///me posiciono en histograma asociado a este bloque y en la posición correspondiente
     atomicAdd((H +index), 1); 
   }
     
 }
/**
 * CUDA Kernel Device code
 * Calcula el histograma de un vector pasado.
 * Versión directa con operaciones atómicas a memoria compartida por un bloque y un solo acceso atómico final, tras sincronización de los hilos
 * con el resultado.
 * a memorida de video.
 * Devuelve un puntero hacia el el vector del histograma ya calculado
 */
int * calculateHistogramByGpu(int * vector,int numElementsV, int numElementsH, bool byBlock, int threadsPerBlock)
{
	size_t sizeH,sizeV;
	int *h_H = NULL;
	int *d_V = NULL;
	int *d_H = NULL;	
    hipError_t err = hipSuccess;
	
	sizeV =  numElementsV * sizeof(int);

	if (threadsPerBlock >1024)
	    threadsPerBlock = 1024;  ///para no sobrepasar el límite de bloque. Realmente este valor se deberá de obtener de la función CUDA adecuada, puesto que podría no ser
							    		
    int blocksPerGrid = (numElementsV + threadsPerBlock - 1) / threadsPerBlock;
	
 if (byBlock)
	///si creo la versión de un histograma por bloque	
	sizeH = numElementsH * blocksPerGrid * sizeof(int) ;
 else
   ///si creo la versión de un histograma único para todo el vector.   
	sizeH = numElementsH * sizeof(int);

 	h_H = (int *)malloc(sizeH);
	
    // Verify that allocations succeeded
    if (h_H == NULL)
    {
        fprintf(stderr, "Failed to allocate host vector Histograma!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vector H a cero
    printf("\nVector Histrograma Inicializado con 0:");
    for (int i = 0; i < (sizeH / sizeof(int)); ++i)    
		h_H[i] = 0;
	
        
    // Allocate the device vector V       
    err = hipMalloc((void **)&d_V, sizeV);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector V (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

// Allocate the device vector H    

    err = hipMalloc((void **)&d_H, sizeH);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector H (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // Copy the host input vectors V in host memory to the device input vectors in
    // device memory
    printf("\nCopy input data from the host memory vector V to the CUDA device");
    err = hipMemcpy(d_V, vector, sizeV, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector V from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors H in host memory to the device input vectors in
    // device memory
    printf("\nCopy input data from the host memory vector H to the CUDA device");
    err = hipMemcpy(d_H, h_H, sizeH, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector H from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	// Launch the Vector Add CUDA Kernel   
	if (byBlock)	
	{
	    printf("\nCUDA kernel - histogramByBlock - launch with %d blocks of %d threads", blocksPerGrid, threadsPerBlock);       
        histogramByBlock<<<blocksPerGrid, threadsPerBlock>>>(d_V, d_H, numElementsV, numElementsH);
        
        if ((blocksPerGrid % 2) !=  0) //si los bloques no son pares, 
            printf("\nBloques impares, todavía no implementada solución, puede fallar");       

        while (blocksPerGrid > 1)
        {
            
            sumHistogram<<<blocksPerGrid /2 , numElementsH>>>(d_H, blocksPerGrid);
            blocksPerGrid /= 2;
            
        }
            

	}
	else
	{
		printf("\nCUDA kernel -histogram- launch with %d blocks of %d threads", blocksPerGrid, threadsPerBlock);       
        //histogram<<<blocksPerGrid, threadsPerBlock>>>(d_V, d_H, numElementsV, numElementsH);        
        histogramShared<<<blocksPerGrid, threadsPerBlock>>>(d_V, d_H, numElementsV, numElementsH);        
	
	}
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	
    // Copy the device result vector V in device memory to the host result vector
    // in host memory.
    printf("\nCopy output data from the CUDA device vector V to the host memory");
    err = hipMemcpy(vector, d_V, sizeV, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector V from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // Copy the device result vector H in device memory to the host result vector
    // in host memory.
    printf("\nCopy output data from the CUDA device vector H to the host memory");
    err = hipMemcpy(h_H, d_H, sizeH, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector H from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // Free device global memory
    err = hipFree(d_V);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector V (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free device global memory
    err = hipFree(d_H);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector V (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


return h_H;
}


/**
 * Calcula el histograma de un vector pasado.

 */
int * calculateHistogramByCpu(int * vector, int numElementsV, int numElementsH)
{
////Version calculo histograma por CPU.     

int * h_H = (int *)malloc(numElementsH * sizeof(int));
  
// Verify that allocations succeeded
if (h_H == NULL)
{
    fprintf(stderr, "Failed to allocate host vector Histograma!\n");
    exit(EXIT_FAILURE);
}

unsigned t0,t1;

t0 = clock();

///Calculo el tiempo que tardaría si se hiciera por CPU
   for (int i = 0; i < numElementsV; i++)
      h_H[vector[i] % numElementsH] = h_H[vector[i] % numElementsH] +1;
     
 t1 = clock();
      double time =  (double (t1-t0)/CLOCKS_PER_SEC);
    
  
///Show Vector H
printf("\nTiempo empleado en calculo por CPU :  %f segundos",time);

return h_H ;
}

/**
 * Host main routine
 */
int main(void)
{

	hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElementsV = 33554432;
	int numElementsH = 8;
	int threadsPerBlock = 1024;
	    
    printf("\nVector V de %d elementos", numElementsV);
    // Allocate the host input vector V
    int *h_V = (int *)malloc(numElementsV * sizeof(int));

    // Verify that allocations succeeded
    if (h_V == NULL)
    {
        fprintf(stderr, "Failed to allocate host vector V!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vector V
    printf("\nVector V Inicializado con :");

    for (int i = 0; i < numElementsV; ++i)    
	 {
		//	h_V[i] = rand();  ///(float)RAND_MAX;
			h_V[i] = i;
		if  (numElementsV<1025)
			printf("\n[%d]", h_V[i]); ///solo muestro por pantalla si es menor o igual de 1024
     }	
     
     


////Version calculo histograma por CPU.     
   
int * h_H = calculateHistogramByCpu(h_V, numElementsV, numElementsH);

    
///Show Vector H
printf("\nResultado Vector Histograma Calculado por CPU ");
for (int i = 0; i < numElementsH; ++i)    
        printf("\n[%d]", h_H[i]);

        
free(h_H);
    
    
    
    h_H = calculateHistogramByGpu(h_V, numElementsV, numElementsH, false, threadsPerBlock);

///Show Vector H
    printf("\nResultado Vector Histograma  :");
    for (int i = 0; i < numElementsH; ++i)    
			printf("\n[%d]", h_H[i]);

free(h_H);

     h_H = calculateHistogramByGpu(h_V, numElementsV, numElementsH, true, threadsPerBlock);   

///Show Vector H
    printf("\nResultado Vector Histograma  :");
    for (int i = 0; i < numElementsH; ++i)    
			printf("\n[%d]", h_H[i]);

free(h_H);





    // Free host memory
    free(h_V);
	

    // Reset the device and exit
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("\nDone\n");
    return 0;
}
